#include "hip/hip_runtime.h"
#include "common.h"
#include <math.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <map>

using namespace std;

//Estructura de nodo
struct Node {
    short x;
    short y;
    short cost;
    float heuristic;
    float score;

    Node() {}

    Node(short x_, short y_, float heuristic_, short cost_) {
        x = x_;
        y = y_;
        heuristic = heuristic_;
        cost = cost_;
        score = heuristic_ + cost_;
    }

    void setNode(short x_, short y_, float heuristic_, short cost_) {
        x = x_;
        y = y_;
        heuristic = heuristic_;
        cost = cost_;
        score = heuristic_ + cost_;
    }

    // Le pasamos un nodo y lo copia a sí
    void copyNode(Node toCopy) {
        x = toCopy.x;
        y = toCopy.y;
        heuristic = toCopy.heuristic;
        cost = toCopy.cost;
        score = toCopy.score;
    }

    void printMe() {
        cout << endl << "x: " << x << endl;
        cout << "y: " << y << endl;
        cout << "heuristic: " << heuristic << endl;
        cout << "cost: " << cost << endl;
        cout << "score: " << score << endl;
    }

    void updateCost(short cost_) {
        cost = cost_;
        score = heuristic + cost_;
    }
};

//Estructura de matriz (maze)
struct Matrix {
    short rows;
    short cols;
    vector<short> data;

    Matrix(short rows_, short cols_):rows(rows_), cols(cols_), data(rows*cols) { }

    short & operator()(size_t row, size_t col) {
        return data[row*cols+col];
    } 

    short operator()(size_t row, size_t col) const {
        return data[row*cols+col];
    } 
};

// Funcion para calcular heuristica de la matriz
__global__ void calcHeuristicOnGPU(float *heuristicMat, short rows, short cols, short finalX, short finalY) {
    //Codigo de clase
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < cols && iy < rows) {
        for(int i = 0; i < rows; i++) {
            // float distance = sqrt(pow(finalX - x, 2) + pow(finalY - y, 2));
            heuristicMat[iy*cols+ix] = sqrt(pow(finalX - ix, 2) + pow(finalY - iy, 2));
            cout << "x: " << ix << " y: " << iy << "heuristic: " << heuristicMat[iy*cols+ix] << endl;
        }
    }
}

//Funcion para hacer el sort
bool sortQueue (Node a, Node b) { return (a.score < b.score); }

string findPath(Node currentNode, map<string, string> &cameFrom) {
    string key = to_string(currentNode.x) + "-" + to_string(currentNode.y);
    string value = cameFrom[key];
    string path = key + "\n";

    while (value != "START") {
        path += value + "\n";
        key = value;
        value = cameFrom[key];
    }
    return path;
}

short isInSet(Node node, vector<Node> &set) {
    for (int i = 0; i < set.size(); i++) {
        if (node.x == set[i].x && node.y == set[i].y) return i;
    }
    return -1;
}

void checkNeighbour(vector<Node> &tempNodes, Matrix maze, short x, short y, short finalX, short finalY, short cost) {
    float distance = sqrt(pow(finalX - x, 2) + pow(finalY - y, 2));
    if (maze(y, x) != 1) {
        tempNodes.push_back(*new Node(x, y, distance, cost));
    }
}

void expandNode(Node currentNode, vector<Node> &openSet, vector<Node> &closedSet, map<string, string> &cameFrom, Matrix maze, short finalX, short finalY) {
    vector<Node> tempNodes;
    short x = currentNode.x;
    short y = currentNode.y;
    short cost = currentNode.cost + 1;

    // Left
    short _x = x - 1;
    short _y = y;
    checkNeighbour(tempNodes, maze, _x, _y, finalX, finalY, cost);
    // Right
    _x = x + 1;
    checkNeighbour(tempNodes, maze, _x, _y, finalX, finalY, cost);
    // Up
    _x = x;
    _y = y - 1;
    checkNeighbour(tempNodes, maze, _x, _y, finalX, finalY, cost);
    // Down
    _y = y + 1;
    checkNeighbour(tempNodes, maze, _x, _y, finalX, finalY, cost);

    // Checamos cada vecino
    for (int i = 0; i < tempNodes.size(); i++) {
        // Si está en el closed set, no hacemos nada con el nodo
        if (isInSet(tempNodes[i], closedSet) > -1) continue;

        short index = isInSet(tempNodes[i], openSet);
        if (index == -1) { // Si no está en openSet lo metemos a openSet
            openSet.push_back(tempNodes[i]);
        } else { // si sí está en openSet, checamos si llegamos con mejor score y lo actualizamos
            if (tempNodes[i].score >= currentNode.score) continue;
            openSet[index].updateCost(tempNodes[i].cost);
        }

        string key = to_string(tempNodes[i].x) + "-" + to_string(tempNodes[i].y);
        string value = to_string(x) + "-" + to_string(y);
        cameFrom[key] = value;
    }
}

void aStarSearch(Matrix maze, short initialX, short initialY, short finalX, short finalY) {
    vector<Node> closedSet; // Set of nodes already evaluated

    //Creamos el nodo inicial
    float heuristic = sqrt(pow(finalX - initialX, 2) + pow(finalY - initialY, 2));
    Node initialNode(initialX, initialY, heuristic, 0);

    // The set of currently discovered nodes that are not evaluated yet.
    // Initially, only the start node is known.
    vector<Node> openSet;
    openSet.push_back(initialNode);

    // For each node, which node it can most efficiently be reached from.
    // If a node can be reached from many nodes, cameFrom will eventually contain the
    // most efficient previous step.
    // cameFrom := an empty map
    // el key es current y el value es parent
    map<string, string> cameFrom;
    string key = to_string(initialNode.x) + "-" + to_string(initialNode.y);
    cameFrom[key] = "START";

    bool foundSoultion = false;
    while(!openSet.empty()) {
        // Sorteamos los nodos dependiendo del score
        sort(openSet.begin(), openSet.end(), sortQueue);
        Node currentNode = openSet.front();

        // Checamos si llegamos al goal
        if (currentNode.x == finalX && currentNode.y == finalY) {
            cout << "solution found" << endl;
            foundSoultion = true;
            ofstream myfile;
            myfile.open ("public/solution.txt");
            myfile << findPath(currentNode, cameFrom);
            myfile.close();
            break;
        }

        move(openSet.begin(), openSet.begin() + 1, back_inserter(closedSet));
        openSet.erase(openSet.begin());
        expandNode(currentNode, openSet, closedSet, cameFrom, maze, finalX, finalY);
    }
    cout << "End of Search" << endl;
}

int main(int argc, char * argv[]) {
    //Tenemos por default el nombre del txt
    char * mazeText = "public/python-generated-maze.txt";

    // Si nos dieron los file names
    if(argc == 2) mazeText = argv[1];

    // Abrimos el file
    FILE* file_ptr = fopen(mazeText, "r");
    
    if(file_ptr == NULL) {
        cout << "ERROR : Unable to open file " << endl;
        exit(EXIT_FAILURE);
    }

    // Inicializamos variables
    short rows, cols, initialX, initialY, finalX, finalY;
    fscanf(file_ptr, "%hu %hu %hu %hu %hu %hu", &rows, &cols, &initialX, &initialY, &finalX, &finalY);

    //Iteramos a traves de la matriz para poner los valores
    Matrix maze(rows, cols);
    for(int i = 0; i < maze.rows; i++) {
        for(int j = 0; j < maze.cols; j++) {
            fscanf(file_ptr, "%hu", &maze(i, j));
        }
    }

    //COSAS DE CUDA -------------------------------------------
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    //Bytes
    short nxy = rows * cols;
    float nBytes = nxy * sizeof(float);

    //MALLOC para host matrix
    float *heuristicMatHost;
    heuristicMatHost = (float *)malloc(nBytes);
    
    //Memset del host matrix
    memset(heuristicMatHost, 0, nBytes);

    // Malloc and copy memory to device
    float *heuristicMat;
    SAFE_CALL(hipMalloc((void **)&heuristicMat, nBytes), "Error allocating heuristicMat");
    // SAFE_CALL(hipMemcpy(heuristicMat, heuristicMatHost, nBytes, hipMemcpyHostToDevice), "Error copying Heuristic Mat to Device");

    // invoke kernel at host side
    int dimx = 16;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

    // Mandamos a llamar a hacer la matriz
    calcHeuristicOnGPU<<<grid, block>>>(heuristicMat, cols, rows, finalX, finalY);

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(heuristicMatHost, heuristicMat, nBytes, hipMemcpyDeviceToHost), "Error copying heuristic back to host");

    // aStarSearch(maze, initialX, initialY, finalX, finalY, heuristicMatHost);

    // free device global memory
    SAFE_CALL(hipFree(heuristicMat), "Error freeing memory");
    
    // free host memory
    free(heuristicMatHost);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return 0;
}